
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CUDA_CHECK_ERROR(X)({\
    if((X) != hipSuccess){\
        fprintf(stderr, "ERROR %d (%s:%d): %s\n", (X), __FILE__, __LINE__, hipGetErrorString((X)));\
        exit(1);\
    }\
})

#define NTHREADS 1024 
#define ALL_THREADS_MASK 0xffffffff
#define WARPSIZE 32


__global__ void vector_reduction_kernel(unsigned char *values, unsigned int nitems, unsigned long long* result){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ unsigned int partial_sums[WARPSIZE];
    unsigned int warpId = threadIdx.x / warpSize;
    unsigned int laneId = threadIdx.x % warpSize; 
    unsigned int myvalue = 0; 
    if(idx < nitems){ 
        myvalue = values[idx]; 
    }
 
    // step 1
    for(unsigned int i = warpSize/2; i >= 1; i /= 2){
        unsigned int up = __shfl_down_sync(ALL_THREADS_MASK, myvalue, i, warpSize); 
        if(laneId < i){
            myvalue += up; 
        }
    }
    if(laneId == 0 && warpId > 0) partial_sums[warpId] = myvalue;
    __syncthreads();
    // step 2
    if(warpId == 0){
        if(laneId > 0) myvalue = partial_sums[laneId];
        for(unsigned int i = warpSize/2; i >= 1; i /= 2){
            unsigned int up = __shfl_down_sync(ALL_THREADS_MASK, myvalue, i, warpSize); 
                if(laneId < i){
                    myvalue += up; 
                }
            }
        if(laneId == 0) atomicAdd(result, myvalue);
    }  
    
}



int main(int argc, char **argv){
    
    unsigned int nitems = 1e9; 
    unsigned char *values = (unsigned char*) malloc(sizeof(unsigned int) * nitems);
    if(!values){
        fprintf(stderr, "Error while allocating memory\n");
        return EXIT_FAILURE;
    }
    // Initialise the vector of n elements to random values
    unsigned long long correct_result = 0;
    for(int i = 0; i < nitems; i++){
        values[i] = (i + 1) % 128;
        correct_result += values[i];
    }
    unsigned long long sum = 0ull;
    unsigned long long *dev_sum;
    unsigned char *dev_values;
    CUDA_CHECK_ERROR(hipMalloc(&dev_values, sizeof(unsigned char) * nitems));
    CUDA_CHECK_ERROR(hipMalloc(&dev_sum, sizeof(unsigned long long)));
    CUDA_CHECK_ERROR(hipMemset(dev_sum, 0, sizeof(unsigned long long)));
    CUDA_CHECK_ERROR(hipMemcpy(dev_values, values, sizeof(unsigned char) * nitems, hipMemcpyHostToDevice));
    unsigned int nblocks = (nitems + NTHREADS - 1) / NTHREADS;
    printf("Number of cuda blocks: %u\n", nblocks);
    hipEvent_t start, stop;
    CUDA_CHECK_ERROR(hipEventCreate(&start));
    CUDA_CHECK_ERROR(hipEventCreate(&stop));
    CUDA_CHECK_ERROR(hipEventRecord(start)); 
    vector_reduction_kernel<<<nblocks, NTHREADS>>>(dev_values, nitems, dev_sum);
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipEventRecord(stop)); 
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    CUDA_CHECK_ERROR(hipMemcpy(&sum, dev_sum, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    float time_spent;
    CUDA_CHECK_ERROR(hipEventElapsedTime(&time_spent, start, stop));
    printf("Result: %llu - Time elapsed: %f\n", sum, time_spent/1000.0f);
    if(correct_result != sum) {
        fprintf(stderr, "Error: sum is not correct, should be %llu\n", correct_result);
        return EXIT_FAILURE;
    }
    return EXIT_SUCCESS;

}
