
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


#define CUDA_CHECK_ERROR(X)({\
    if((X) != hipSuccess){\
        fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, hipGetErrorString((X)));\
        exit(1);\
    }\
})



int main(int argc, char** argv){
	int count;			// variable for number of devices
	int device;			// variable for active device id

	CUDA_CHECK_ERROR(hipGetDeviceCount(&count));
	printf("\nFound %i devices\n\n", count);

	for (device = 0; device < count; device++){
		CUDA_CHECK_ERROR(hipSetDevice(device));

		struct hipDeviceProp_t p;
		CUDA_CHECK_ERROR(hipGetDeviceProperties(&p, device));

		printf("Device %i : ", device);
		printf("%s ", p.name);
		printf("with %i SMs\n", p.multiProcessorCount);
	}
	
	printf("\n");
	return 0;
}
