
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


void __cuda_check_error(hipError_t err, const char *file, int line){
	if(err != hipSuccess){
        fprintf(stderr, "CUDA error (%s:%d): %s\n", file, line, hipGetErrorString(err));
        exit(1);
    }
}


#define CUDA_CHECK_ERROR(X)({\
	__cuda_check_error((X), __FILE__, __LINE__);\
})

#define NTHREADS 1024 



__global__ void vector_reduction_kernel(unsigned char *values, unsigned int nitems, unsigned long long* result){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < nitems) atomicAdd(result, values[idx]);  
}



int main(int argc, char **argv){
    
    unsigned int nitems = 1e6; 
    unsigned char *values = (unsigned char*) malloc(sizeof(unsigned int) * nitems);
    if(!values){
        fprintf(stderr, "Error while allocating memory\n");
        return EXIT_FAILURE;
    }
    // Initialise the vector of n elements to random values
    unsigned long long correct_result = 0;
    for(int i = 0; i < nitems; i++){
        values[i] = (i + 1) % 128;
        correct_result += values[i];
    }

    unsigned long long sum = 0ull;
    unsigned long long *dev_sum;
    unsigned char *dev_values;
    CUDA_CHECK_ERROR(hipMalloc(&dev_values, sizeof(unsigned char) * nitems));
    CUDA_CHECK_ERROR(hipMalloc(&dev_sum, sizeof(unsigned long long)));
    CUDA_CHECK_ERROR(hipMemset(dev_sum, 0, sizeof(unsigned long long)));
    CUDA_CHECK_ERROR(hipMemcpy(dev_values, values, sizeof(unsigned char) * nitems, hipMemcpyHostToDevice));
    unsigned int nblocks = (nitems + NTHREADS - 1) / NTHREADS;
    hipEvent_t start, stop;
    CUDA_CHECK_ERROR(hipEventCreate(&start));
    CUDA_CHECK_ERROR(hipEventCreate(&stop));
    CUDA_CHECK_ERROR(hipEventRecord(start)); 
    vector_reduction_kernel<<<nblocks, NTHREADS>>>(dev_values, nitems, dev_sum);
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipEventRecord(stop)); 
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    CUDA_CHECK_ERROR(hipMemcpy(&sum, dev_sum, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    float time_spent;
    CUDA_CHECK_ERROR(hipEventElapsedTime(&time_spent, start, stop));
    printf("Result: %llu - Time elapsed: %f\n", sum, time_spent/1000.0f);
    if(correct_result != sum) {
        fprintf(stderr, "Error: sum is not correct, should be %llu\n", correct_result);
        return EXIT_FAILURE;
    }
    return EXIT_SUCCESS;

}
