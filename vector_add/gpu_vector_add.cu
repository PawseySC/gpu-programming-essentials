
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



void __cuda_check_error(hipError_t err, const char *file, int line){
	if(err != hipSuccess){
        fprintf(stderr, "CUDA error (%s:%d): %s\n", file, line, hipGetErrorString(err));
        exit(1);
    }
}


#define CUDA_CHECK_ERROR(X)({\
	__cuda_check_error((X), __FILE__, __LINE__);\
})



#define MALLOC_CHECK_ERROR(X)({\
    if ((X) == 0){\
        fprintf(stderr, "Malloc error (%s:%d): %i\n", __FILE__, __LINE__, (X));\
        exit(1);\
    }\
})


// Returns True if |a - b| <= eps
inline bool compare_float(float a, float b){
    const float eps = 1e-7f;
    if (a  > b) return a - b <= eps;
    else return b - a <= eps;
}



// Initialise the vector v of n elements to random values
void init_vec(float *v, int n){
    for(int i = 0; i < n; i++){
        v[i] = rand() % 100 * 0.3234f;
    }
}


// kernel to perform vector addition
__global__ void vector_add(float *a, float *b, float *c, int n){
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}



int main(void){
    const int n = 2000;
    float *A = (float*) malloc(n * sizeof(float));
    float *B = (float*) malloc(n * sizeof(float));
    float *C = (float*) malloc(n * sizeof(float));
    MALLOC_CHECK_ERROR(A && B && C);
    init_vec(A, n);
    init_vec(B, n);
    float *dev_A, *dev_B, *dev_C;
    CUDA_CHECK_ERROR(hipMalloc(&dev_A, sizeof(float) * n));
    CUDA_CHECK_ERROR(hipMalloc(&dev_B, sizeof(float) * n));
    CUDA_CHECK_ERROR(hipMalloc(&dev_C, sizeof(float) * n));
    CUDA_CHECK_ERROR(hipMemcpy(dev_A, A, sizeof(float) * n, hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(dev_B, B, sizeof(float) * n, hipMemcpyHostToDevice));
    const int nThreads = 1024;
    const int nBlocks = (n + nThreads - 1) / nThreads;
    vector_add<<<nBlocks, 1025>>>(dev_A, dev_B, dev_C, n);
    CUDA_CHECK_ERROR((hipError_t)hipGetLastError());
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    CUDA_CHECK_ERROR(hipMemcpy(C, dev_C, sizeof(float) * n, hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    
    // check the result is correct
    for(int i = 0; i < n; i++){
        bool sums_equal = compare_float(C[i], A[i] + B[i]);
        if(!sums_equal){
            fprintf(stderr, "Sum is not correct.\n");
            hipFree(dev_A);
            hipFree(dev_B);
            hipFree(dev_C);
            free(A);
            free(B);
            free(C);
            return 1;
        }
    }
    CUDA_CHECK_ERROR(hipFree(dev_A));
    CUDA_CHECK_ERROR(hipFree(dev_B));
    CUDA_CHECK_ERROR(hipFree(dev_C));
    free(A);
    free(B);
    free(C);
    printf("All good.\n");
    return 0;
}
