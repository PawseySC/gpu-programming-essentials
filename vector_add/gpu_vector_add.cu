
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



#define CUDA_CHECK_ERROR(X)({\
    if((X) != hipSuccess){\
        fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, hipGetErrorString((X)));\
        exit(1);\
    }\
})



#define MALLOC_CHECK_ERROR(X)({\
    if ((X) == 0){\
        fprintf(stderr, "Malloc error (%s:%d): %i\n", __FILE__, __LINE__, (X));\
        exit(1);\
    }\
})


// Returns True if |a - b| <= eps
inline bool compare_float(float a, float b){
    const float eps = 1e-7f;
    if (a  > b) return a - b <= eps;
    else return b - a <= eps;
}



// Initialise the vector v of n elements to random values
void init_vec(float *v, int n){
    for(int i = 0; i < n; i++){
        v[i] = rand() % 100 * 0.3234f;
    }
}


// kernel to perform vector addition
__global__ void vector_add(float *a, float *b, float *c, int n){
    unsigned int i = threadIdx.x;
    c[i] = a[i] + b[i];
}



int main(void){
    int n = 100;
    float *A = (float*) malloc(n * sizeof(float));
    float *B = (float*) malloc(n * sizeof(float));
    float *C = (float*) malloc(n * sizeof(float));
    MALLOC_CHECK_ERROR(A && B && C);
    init_vec(A, n);
    init_vec(B, n);
    float *dev_A, *dev_B, *dev_C;
    CUDA_CHECK_ERROR(hipMalloc(&dev_A, sizeof(float) * n));
    CUDA_CHECK_ERROR(hipMalloc(&dev_B, sizeof(float) * n));
    CUDA_CHECK_ERROR(hipMalloc(&dev_C, sizeof(float) * n));
    CUDA_CHECK_ERROR(hipMemcpy(dev_A, A, sizeof(float) * n, hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(dev_B, B, sizeof(float) * n, hipMemcpyHostToDevice));
    vector_add<<<1, n>>>(dev_A, dev_B, dev_C, n);
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipMemcpy(C, dev_C, sizeof(float) * n, hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    
    // check the result is correct
    for(int i = 0; i < n; i++){
        bool sums_equal = compare_float(C[i], A[i] + B[i]);
        if(!sums_equal){
            fprintf(stderr, "Sum is not correct.\n");
            hipFree(dev_A);
            hipFree(dev_B);
            hipFree(dev_C);
            free(A);
            free(B);
            free(C);
            return 1;
        }
    }
    CUDA_CHECK_ERROR(hipFree(dev_A));
    CUDA_CHECK_ERROR(hipFree(dev_B));
    CUDA_CHECK_ERROR(hipFree(dev_C));
    free(A);
    free(B);
    free(C);
    printf("All good.\n");
    return 0;
}
