
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CUDA_CHECK_ERROR(X)({\
    if((X) != hipSuccess){\
        fprintf(stderr, "ERROR %d (%s:%d): %s\n", (X), __FILE__, __LINE__, hipGetErrorString((X)));\
        exit(1);\
    }\
})

#define NTHREADS 1024 



__global__ void vector_sum(unsigned char *values, unsigned int nitems, unsigned long long* result){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ unsigned int partial_sums[32];
    unsigned int warpId = threadIdx.x / 32;
    unsigned int laneId = threadIdx.x % 32; 
    unsigned int gridSize = gridDim.x * blockDim.x;
    unsigned int nloops = (nitems + gridSize  - 1) / gridSize;
    unsigned int l = 0;
    unsigned int myvalue;

    while(l++ < nloops){
        if(idx < nitems){ 
            myvalue = values[idx]; 
        }else{
            myvalue = 0;
        }
 
        // step 1
        for(unsigned int i = 16; i >= 1; i >>= 1){
            unsigned int up = __shfl_down_sync(0xffffffff, myvalue, i, 32); 
            if(laneId < i){
                myvalue += up; 
            }
        }
        if(laneId == 0 && warpId > 0) partial_sums[warpId] = myvalue;
       
        __syncthreads();
        // step 2
        if(warpId == 0){
            if(laneId > 0) myvalue = partial_sums[laneId];
            for(unsigned int i = 16; i >= 1; i >>= 1){
                unsigned int up = __shfl_down_sync(0xffffffff, myvalue, i, 32); 
                    if(laneId < i){
                        myvalue += up; 
                    }
                }
            if(laneId == 0) atomicAdd(result, myvalue);
        }  
        idx += gridSize;
        __syncthreads();
    }
}



int main(int argc, char **argv){
    
    unsigned int nitems = 1e9; 
    unsigned char *values = (unsigned char*) malloc(sizeof(unsigned int) * nitems);
    if(!values){
        fprintf(stderr, "Error while allocating memory\n");
        return -1;
    }
    // Initialise the vector of n elements to random values
    unsigned long long correct_result = 0;
    for(int i = 0; i < nitems; i++){
        values[i] = rand() % 256;
        correct_result += values[i];
    }
    unsigned long long sum = 0ull;
    unsigned long long *dev_sum;
    unsigned char *dev_values;
    CUDA_CHECK_ERROR(hipMalloc(&dev_values, sizeof(unsigned char) * nitems));
    CUDA_CHECK_ERROR(hipMalloc(&dev_sum, sizeof(unsigned long long)));
    CUDA_CHECK_ERROR(hipMemset(dev_sum, 0, sizeof(unsigned long long)));
    CUDA_CHECK_ERROR(hipMemcpy(dev_values, values, sizeof(unsigned char) * nitems, hipMemcpyHostToDevice));
    struct hipDeviceProp_t props;
    CUDA_CHECK_ERROR(hipGetDeviceProperties(&props, 0));
    unsigned int nblocks = props.multiProcessorCount * 2;
    printf("Number of cuda blocks: %u\n", nblocks);
    hipEvent_t start, stop;
    CUDA_CHECK_ERROR(hipEventCreate(&start));
    CUDA_CHECK_ERROR(hipEventCreate(&stop));
    CUDA_CHECK_ERROR(hipEventRecord(start)); 
    vector_sum<<<nblocks, NTHREADS>>>(dev_values, nitems, dev_sum);
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipEventRecord(stop)); 
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    CUDA_CHECK_ERROR(hipMemcpy(&sum, dev_sum, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    float time_spent;
    CUDA_CHECK_ERROR(hipEventElapsedTime(&time_spent, start, stop));
    printf("Result: %llu - Time elapsed: %f\n", sum, time_spent/1000.0f);
    if(correct_result != sum) {
        fprintf(stderr, "Error: sum is not correct, should be %llu\n", correct_result);
        return 1;
    }
    return 0;

}
