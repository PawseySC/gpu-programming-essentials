
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CUDA_CHECK_ERROR(X)({\
    if((X) != hipSuccess){\
        fprintf(stderr, "ERROR %d (%s:%d): %s\n", (X), __FILE__, __LINE__, hipGetErrorString((X)));\
        exit(1);\
    }\
})

#define NTHREADS 1024 



__global__ void vector_sum(unsigned char *values, unsigned int nitems, unsigned long long* result){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < nitems) atomicAdd(result, values[idx]);  
}



int main(int argc, char **argv){
    
    unsigned int nitems = 1e9; 
    unsigned char *values = (unsigned char*) malloc(sizeof(unsigned int) * nitems);
    if(!values){
        fprintf(stderr, "Error while allocating memory\n");
        return -1;
    }
    // Initialise the vector of n elements to random values
    unsigned long long correct_result = 0;
    for(int i = 0; i < nitems; i++){
        values[i] = rand() % 256;
        correct_result += values[i];
    }

    unsigned long long sum = 0ull;
    unsigned long long *dev_sum;
    unsigned char *dev_values;
    CUDA_CHECK_ERROR(hipMalloc(&dev_values, sizeof(unsigned char) * nitems));
    CUDA_CHECK_ERROR(hipMalloc(&dev_sum, sizeof(unsigned long long)));
    CUDA_CHECK_ERROR(hipMemset(dev_sum, 0, sizeof(unsigned long long)));
    CUDA_CHECK_ERROR(hipMemcpy(dev_values, values, sizeof(unsigned char) * nitems, hipMemcpyHostToDevice));
    unsigned int nblocks = (nitems + NTHREADS - 1) / NTHREADS;
    hipEvent_t start, stop;
    CUDA_CHECK_ERROR(hipEventCreate(&start));
    CUDA_CHECK_ERROR(hipEventCreate(&stop));
    CUDA_CHECK_ERROR(hipEventRecord(start)); 
    vector_sum<<<nblocks, NTHREADS>>>(dev_values, nitems, dev_sum);
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipEventRecord(stop)); 
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    CUDA_CHECK_ERROR(hipMemcpy(&sum, dev_sum, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    float time_spent;
    CUDA_CHECK_ERROR(hipEventElapsedTime(&time_spent, start, stop));
    printf("Result: %llu - Time elapsed: %f\n", sum, time_spent/1000.0f);
    if(correct_result != sum) {
        fprintf(stderr, "Error: sum is not correct, should be %llu\n", correct_result);
        return 1;
    }
    return 0;

}
