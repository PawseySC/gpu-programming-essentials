
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>



#define CUDA_CHECK_ERROR(X)({\
    if((X) != hipSuccess){\
        fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, hipGetErrorString((X)));\
        exit(1);\
    }\
})


// Initialise the vector v of n elements to random values
void init_vec(float *v, int n){
    for(int i = 0; i < n; i++){
        v[i] = rand() % 100 * 0.3234f;
    }
}


// kernel to perform vector addition
__global__ void vector_add(float *a, float *b, float *c, int n){
    unsigned int i = threadIdx.x;
    c[i] = a[i] + b[i];
}



int main(void){
    int n = 100;
    float *A = new float[n];
    float *B = new float[n];
    float *C = new float[n];
    init_vec(A, n);
    init_vec(B, n);
    float *dev_A, *dev_B, *dev_C;
    CUDA_CHECK_ERROR(hipMalloc(&dev_A, sizeof(float) * n));
    CUDA_CHECK_ERROR(hipMalloc(&dev_B, sizeof(float) * n));
    CUDA_CHECK_ERROR(hipMalloc(&dev_C, sizeof(float) * n));
    CUDA_CHECK_ERROR(hipMemcpy(dev_A, A, sizeof(float) * n, hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(dev_B, B, sizeof(float) * n, hipMemcpyHostToDevice));
    vector_add<<<1, n>>>(dev_A, dev_B, dev_C, n);
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipMemcpy(C, dev_C, sizeof(float) * n, hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    
    // check the result is correct
    for(int i = 0; i < n; i++){
        if(C[i] != A[i] + B[i]){
            std::cerr << "Sum is not correct." << std::endl;
            hipFree(dev_A);
            hipFree(dev_B);
            hipFree(dev_C);
            delete[] A;
            delete[] B;
            delete[] C;
            return 1;
        }
    }
    CUDA_CHECK_ERROR(hipFree(dev_A));
    CUDA_CHECK_ERROR(hipFree(dev_B));
    CUDA_CHECK_ERROR(hipFree(dev_C));
    delete[] A;
    delete[] B;
    delete[] C;
    std::cout << "All good." << std::endl;
    return 0;
}
